#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include <time.h>
#include <hiprand/hiprand_kernel.h>


extern double* occluded_image;
extern int* occluded_mask;

extern double *train_images;

extern double *d_means;

extern double* test_images;
extern double* dist;

extern "C" void createData(size_t length, size_t cols, size_t test_length){
    hipMallocManaged( &occluded_image, ( test_length * cols * sizeof(double))); 
    hipMemset(occluded_image, 0, test_length * cols * sizeof(double));

    hipMallocManaged( &occluded_mask, ( test_length * cols * sizeof(int))); 
    hipMemset(occluded_mask, 0, test_length * cols * sizeof(int));

    hipMallocManaged( &dist, ( length * cols *sizeof(double))); 
    hipMemset(dist, 0, length * cols *sizeof(double));

    hipMallocManaged( &d_means, ( cols *sizeof(double))); 
    hipMemset(d_means, 0, cols *sizeof(double));

    hipMallocManaged( &train_images, ( length * cols * sizeof(double)));
    hipMemset(train_images, 0, length * cols * sizeof(double));

    hipMallocManaged( &test_images, ( test_length * cols * sizeof(double))); 
    hipMemset(test_images, 0, test_length * cols * sizeof(double));
}

// __global__ void occludedKernel(double* data, double* test_images, int* occluded_mask_arg, size_t rows, size_t cols) {
//     extern __shared__ int shared_mask[];
//     size_t i = threadIdx.x;
//     for (; i < cols; i += blockDim.x) {
//         shared_mask[i] = occluded_mask_arg[blockIdx.x * blockDim.x + i];
//     }
//     __syncthreads();
//     size_t start = (blockIdx.x * blockDim.x + threadIdx.x) / cols;
//     for(size_t idx = threadIdx.x; idx < cols; idx += blockDim.x) {
//         start *= cols;
//         int width = sqrt((float) cols);
//         if(shared_mask[idx] == 1){
//             int left = idx - 1;
//             while(left > 0 && left % width != (width - 1) && shared_mask[left] == 1) left--;
//             int right = idx + 1;
//             while(right < cols - 1 && right % width != 0 && shared_mask[right] == 1) right++;
//             int top = idx - width;
//             while(top > 0 && shared_mask[top] == 1) top-= width;
//             int bottom = idx + width;
//             while(bottom < cols - 1 && shared_mask[bottom] == 1) bottom+= width;

//             double interpolated_value = 0.0;
//             int count = 0;
//             if(left >= 0 && shared_mask[left] == 0){
//                 interpolated_value += test_images[blockIdx.x * blockDim.x + left];
//                 count++;
//             }
//             if(right < cols && shared_mask[right] == 0){
//                 interpolated_value += test_images[blockIdx.x * blockDim.x + right];
//                 count++;
//             }
//             if(top >= 0 && shared_mask[top] == 0){
//                 interpolated_value += test_images[blockIdx.x * blockDim.x + top];
//                 count++;
//             }
//             if(bottom < cols && shared_mask[bottom] == 0){
//                 interpolated_value += test_images[blockIdx.x * blockDim.x + bottom];
//                 count++;
//             }
//             if(count > 0){
//                 data[blockIdx.x * blockDim.x + idx] = interpolated_value / count;
//                 data[0] = 12;
//             }
//         }else{
//             data[blockIdx.x * blockDim.x + idx] = test_images[blockIdx.x * blockDim.x + idx];
//             data[0] = 11;
//         }
//     }
// }

__global__ void occludedKernel(double* data, double* test_images, int* occluded_mask_arg, size_t start_row, size_t rows, size_t cols) {
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < cols * rows; index += blockDim.x * gridDim.x) {
        size_t idx = index + start_row * cols;
        size_t start = (int) idx / cols;
        start *= cols;
        int width = sqrt((float) cols);
        if(occluded_mask_arg[idx] == 1){
            int left = idx - 1;
            while(left > start && left % width != (width - 1) && occluded_mask_arg[left] == 1) left--;
            int right = idx + 1;
            while(right < start + width && right % width != 0 && occluded_mask_arg[right] == 1) right++;
            int top = idx - width;
            while(top > start && occluded_mask_arg[top] == 1) top-= width;
            int bottom = idx + width;
            while(bottom < start + cols && occluded_mask_arg[bottom] == 1) bottom+= width;

            double interpolated_value = 0.0;
            int count = 0;
            if(left >= start && left % width != (width - 1) && occluded_mask_arg[left] == 0){
                interpolated_value += test_images[left];
                count++;
            }
            if(right < start + width && right % width != 0 && occluded_mask_arg[right] == 0){
                interpolated_value += test_images[right];
                count++;
            }
            if(top >= start && occluded_mask_arg[top] == 0){
                interpolated_value += test_images[top];
                count++;
            }
            if(bottom < start + cols && occluded_mask_arg[bottom] == 0){
                interpolated_value += test_images[bottom];
                count++;
            }
            if(count > 0){
                data[idx] = interpolated_value / count;
            }
        }else{
            data[idx] = test_images[idx]; 
        }
    }
}

extern "C" void occludedKernelLaunch(double** data, double* test_images, int* occluded_mask_arg, size_t start, size_t rows, size_t cols, size_t threadsCount){
    size_t blockSize = threadsCount;
    size_t gridSize = (((rows * cols) + blockSize - 1) / blockSize);
    occludedKernel<<<gridSize, blockSize>>>(*data, test_images, occluded_mask_arg, start, rows, cols);
    hipDeviceSynchronize();
}


__global__ void createMaskKernel(int* data, size_t start, size_t rows, size_t cols){
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < cols * rows; index += blockDim.x * gridDim.x) {
        size_t idx = index + start * cols;
        hiprandState state;
        hiprand_init(123, idx, 0, &state);
        //Performance won't vary, if same test size is used
        data[idx] = (hiprand_uniform(&state) < 0.5) ? 0 : 1;
    }
}

extern "C" void createMask(int** data, size_t start, size_t rows, size_t cols, size_t threadsCount){
    size_t blockSize = threadsCount;
    size_t gridSize = (((rows * cols) + blockSize - 1) / blockSize);
    createMaskKernel<<<gridSize, blockSize>>>(*data, start, rows, cols);
    hipDeviceSynchronize();
}


__global__ void normalizeKernel(double* data, size_t rows, size_t cols){
    for(size_t row_idx = blockIdx.x * blockDim.x + threadIdx.x; row_idx < rows; row_idx += blockDim.x * gridDim.x) {
        int start = row_idx * cols;
        double sum = 0.0;

        // Calculate the sum of squares of elements in the row
        for (int i = start; i < start + cols; i++) {
            sum += data[i] * data[i];
        }

        // Normalize each element in the row by dividing by the square root of the sum of squares
        double norm_factor = sqrt(sum);
        for (int i = start; i < start + cols; i++) {
            data[i] /= norm_factor;
        }
    }
}

extern "C" void normalizeKernelLaunch(double** data, size_t rows, size_t cols, size_t threadsCount){
    size_t blockSize = threadsCount;
    size_t gridSize = (((rows * cols) + blockSize - 1) / blockSize);
    normalizeKernel<<<gridSize, blockSize>>>(*data, rows, cols);
    hipDeviceSynchronize();
}


__global__ void meanCenterKernel(double* data, double* means, size_t rows, size_t cols){
    for(size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < rows * cols; idx += blockDim.x * gridDim.x) {
        size_t loc = idx % cols;
        data[idx] -= means[loc];
    }
}


extern "C" void meanCenterKernelLaunch(double** data, double* means, size_t rows, size_t cols, size_t threadsCount){
    size_t blockSize = threadsCount;
    size_t gridSize = (((rows * cols) + blockSize - 1) / blockSize);
    // cudaMemcpy(d_means, *means, cols*sizeof(double), cudaMemcpyHostToDevice);
    meanCenterKernel<<<gridSize, blockSize>>>(*data, means, rows, cols);
    hipDeviceSynchronize();
}

// Not using shared memory because of critical section needing mutexes
__global__ void norm2Kernel(double* images, double* test, size_t rows, size_t cols, size_t start_test, double* answer){
    size_t index;
    for(index = blockIdx.x * blockDim.x + threadIdx.x; index < rows; index += blockDim.x * gridDim.x) {
        double sum = 0.0;
        size_t start = index * cols;
        for(int i = start; i < cols + start; i++){
            sum += (images[i] - test[start_test + i - start]) * (images[i] - test[start_test + i - start]);
        }
        answer[index] = sqrt(sum);
    }
}

extern "C" void norm2KernelLaunch(double** data, double* test, size_t rows, size_t cols, size_t start, size_t threadsCount, double** answer){
    size_t blockSize = threadsCount;
    size_t gridSize = (((rows * cols) + blockSize - 1) / blockSize);
    norm2Kernel<<<gridSize, blockSize>>>(*data, test, rows, cols, start, *answer);
    hipDeviceSynchronize();
}

extern "C" void freeData(){
    hipFree(occluded_image);
    hipFree(occluded_mask);
    hipFree(train_images);
    hipFree(test_images);
    hipFree(dist);
    hipFree(d_means);
}